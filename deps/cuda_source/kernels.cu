#include <hip/hip_runtime.h>

__global__ void add_kernel(const float *a, const float *b, float *c, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    c[idx] = a[idx] + b[idx];
  }
}

// This is the C-callable wrapper function that we will call from Julia.
extern "C" void add_vectors(const float *a, const float *b, float *c, int n) {
  int threads_per_block = 256;
  int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;

  add_kernel<<<blocks_per_grid, threads_per_block>>>(a, b, c, n);

  // It's good practice to check for errors, though we'll keep it simple here.
  hipDeviceSynchronize();
}
